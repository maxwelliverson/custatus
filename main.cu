#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <execution>

#include "event.h"
#include "status.h"

class memory_resource{
  virtual void* do_allocate(size_t Size, size_t Alignment, void* Previous) noexcept = 0;
  virtual void do_deallocate(void* Pointer) noexcept = 0;

  virtual bool do_is_equal(const memory_resource& Other) const noexcept{
    return this == &Other;
  }
public:

};

class allocator{
  void*(*allocate)(size_t Bytes);
  void(*deallocate)(void* Pointer);
};


namespace detail{
  struct address_control_block{

  };
}

template <typename T>
class uniform_address{
  template <typename>
  friend class host_pointer;
  template <typename>
  friend class device_pointer;

  T* DevicePointer;
  T* HostPointer;
public:

};

template <typename T>
class host_pointer{
  T* Pointer;
public:
  T* operator->() const noexcept{
    return Pointer;
  }
  T& operator*() const noexcept{
    return *Pointer;
  }
};
template <typename T>
class device_pointer{
  T* Pointer;
public:
  __device__ T* operator->() const noexcept{
    return Pointer;
  }
  __device__ T& operator*() const noexcept{
    return *Pointer;
  }
};


class error{};
class status_code{
public:
  struct domain{
    cu::string_view(*name)();
    cu::string_view(*message)(int64_t Value);
    bool(*equivalent)(const domain* Other, )
  };

  status_code();

  int64_t value() const noexcept;
  cu::string_view message() const noexcept;

private:
  int64_t Value;
  const domain* Domain;
};




template <typename T>
class owned_ptr{
  struct deleter{

  };
public:
  using pointer = T*;
  using reference = T&;
  using const_pointer = std::conditional_t<std::is_pointer_v<pointer>, pointer, const pointer&>;

  owned_ptr() = default;
  owned_ptr(const_pointer Ptr) noexcept : Ptr(Ptr){}
  owned_ptr(std::nullptr_t) noexcept {}
  owned_ptr(const owned_ptr&) = delete;
  owned_ptr(owned_ptr&& Other) noexcept : Ptr(Other.release()){}

  ~owned_ptr(){

  }

  owned_ptr& operator=(const owned_ptr&) = delete;
  owned_ptr& operator=(owned_ptr&& Other) noexcept{
    if(Other.Ptr != Ptr)
      reset(Other.release());
    return *this;
  }

  reference operator*() const noexcept{
    return *Ptr;
  }
  pointer operator->() const noexcept{
    return Ptr;
  }

  pointer get() const noexcept{
    return Ptr;
  }
  pointer release(){
    pointer Tmp = Ptr;
    Ptr = nullptr;
    return Tmp;
  }
  void reset(pointer Pointer = nullptr){
    (*this)(Ptr);
    Ptr = Pointer;
  }

private:
  pointer Ptr = nullptr;
};
template <typename T>
class borrowed_ptr{
public:
  borrowed_ptr() = default;
  borrowed_ptr(const owned_ptr<T>)
private:
  T* Pointer;
};








class CUioStream{
  char* Buffer;
public:

};




struct memory_range{
  hipDeviceptr_t Address;
  size_t Size;
};

std::ostream& operator<<(std::ostream& OS, hipMemAllocationHandleType HandleTypes){
  OS << "{ ";
  bool Flag = false;
  if(HandleTypes & hipMemHandleTypePosixFileDescriptor) {
    OS << "posix_fd | ";
    Flag = true;
  }
  if(HandleTypes & hipMemHandleTypeWin32) {
    OS << "win32 | ";
    Flag = true;
  }
  if(HandleTypes & hipMemHandleTypeWin32Kmt)
    OS << "win32_kmt ";
  if(Flag)
    OS << "\b\b";
  return OS << "}";
}
std::ostream& operator<<(std::ostream& OS, hipMemoryType MemoryType){
  switch(MemoryType){
    case hipMemoryTypeHost:
      return OS << "host";
    case hipMemoryTypeDevice:
      return OS << "device";
    case hipMemoryTypeArray:
      return OS << "array";
    case hipMemoryTypeUnified:
      return OS << "unified";
    default:
      return OS << "unknown";
  }
}

class ptr_attributes{
  hipCtx_t Ctx;
  hipMemoryType MemoryType;
  hipDeviceptr_t DevicePtr;
  void* HostPtr;
  bool SyncMemops;
  bool IsManaged;
  bool Mapped;
  hipDevice_t Device;
  uint64_t BufferId;
  hipDeviceptr_t RangeStartAddr;
  size_t RangeSize;
  hipMemAllocationHandleType HandleTypes;

  inline static hipPointer_attribute Attributes[]{
      HIP_POINTER_ATTRIBUTE_CONTEXT,
      HIP_POINTER_ATTRIBUTE_MEMORY_TYPE,
      HIP_POINTER_ATTRIBUTE_DEVICE_POINTER,
      HIP_POINTER_ATTRIBUTE_HOST_POINTER,
      HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS,
      HIP_POINTER_ATTRIBUTE_BUFFER_ID,
      HIP_POINTER_ATTRIBUTE_IS_MANAGED,
      HIP_POINTER_ATTRIBUTE_DEVICE_ORDINAL,
      HIP_POINTER_ATTRIBUTE_RANGE_START_ADDR,
      HIP_POINTER_ATTRIBUTE_RANGE_SIZE,
      HIP_POINTER_ATTRIBUTE_MAPPED,
      HIP_POINTER_ATTRIBUTE_ALLOWED_HANDLE_TYPES
  };
public:
  explicit ptr_attributes(hipDeviceptr_t Ptr){
    void* Ptrs[]{
        &Ctx,
        &MemoryType,
        &DevicePtr,
        &HostPtr,
        &SyncMemops,
        &BufferId,
        &IsManaged,
        &Device,
        &RangeStartAddr,
        &RangeSize,
        &Mapped,
        &HandleTypes
    };
    cuda_assert((hipDrvPointerGetAttributes)(std::size(Attributes), Attributes, Ptrs, Ptr))
  }

  [[nodiscard]] hipCtx_t context() const noexcept{
    return Ctx;
  }
  [[nodiscard]] hipMemoryType memory_type() const noexcept{
    return MemoryType;
  }
  [[nodiscard]] hipDeviceptr_t device_address() const noexcept{
    return DevicePtr;
  }
  [[nodiscard]] void* host_address() const noexcept{
    return HostPtr;
  }
  [[nodiscard]] bool sync_mem_ops() const noexcept{
    return SyncMemops;
  }
  [[nodiscard]] uint64_t buffer_id() const noexcept{
    return BufferId;
  }
  [[nodiscard]] bool managed() const noexcept{
    return IsManaged;
  }
  [[nodiscard]] bool mapped() const noexcept{
    return Mapped;
  }
  [[nodiscard]] hipDevice_t device() const noexcept{
    return Device;
  }
  [[nodiscard]] memory_range range() const noexcept{
    return {RangeStartAddr, RangeSize};
  }
  [[nodiscard]] hipMemAllocationHandleType allowed_handle_types() const noexcept{
    return HandleTypes;
  }

  friend std::ostream& operator<<(std::ostream& OS, const ptr_attributes& A){
    OS << std::boolalpha;
    OS << "{\n\tdevice_address: " << (void*)A.DevicePtr << ",\n\thost_address: " << A.HostPtr;
    OS << ",\n\n\tcontext: " << A.Ctx << ",\n\tdevice: " << A.Device << ",\n\n\t";
    OS << "memory_type: " << A.MemoryType << ",\n\tenabled_handles: "<< A.HandleTypes << ",\n\n\t";
    OS << "buffer_id: " << A.BufferId << ",\n\trange: {\n\t\taddress: " << (void*)A.RangeStartAddr << ",\n\t\t";
    OS << "size: " << A.RangeSize << "\n\t},\n\n\t" << "mapped: " << A.Mapped << ",\n\t";
    OS << "managed: " << A.IsManaged <<",\n\tmemory_sync: " << A.SyncMemops << "\n}";
    return OS;
  }
};



inline static constexpr size_t align_to(size_t Size, size_t Align){
  return Size & (Align - 1) ? (Size | (Align - 1)) + 1 : Size;
}


__global__ void twiddle_thumbs(char* Message){
  uint32_t Times = 100;
  while(--Times)
    asm volatile("nanosleep.u32 400000000;");
  if(threadIdx.x == 0){
    unsigned I = 0;
    for(char C : "Hello Maxwell Iverson, this is your GPU. I've slept for 4 seconds now, are you happy??"){
      Message[I++] = C;
    }
  }
}

int main(int argc, char** argv) {

  hipMemGenericAllocationHandle_t Handle;
  hipDeviceptr_t Address;
  size_t RangeSize;
  size_t Alignment;
  hipDevice_t Device;
  hipCtx_t Ctx;
  void* Ptr;

  hipMemAllocationProp AllocationProps;
  AllocationProps.requestedHandleTypes = hipMemHandleTypePosixFileDescriptor;
  AllocationProps.type = hipMemAllocationTypePinned;
  AllocationProps.win32HandleMetaData = nullptr;
  AllocationProps.location.type = hipMemLocationTypeDevice;
  AllocationProps.location.id = 0;
  AllocationProps.allocFlags = {};

  if(argc == 2)
    RangeSize = std::strtoull(argv[1], nullptr, 0);
  else
    std::cin >> RangeSize;


  unsigned ApiVersion;
  int UnifiedAddressing;

  cuda_assert((hipInit)(0))
  cuda_assert((hipDeviceGet)(&Device, 0))
  cuda_assert((hipDevicePrimaryCtxRetain)(&Ctx, Device))
  cuda_assert((hipCtxGetApiVersion)(Ctx, &ApiVersion))
  cuda_assert((hipMemGetAllocationGranularity)(&Alignment, &AllocationProps, hipMemAllocationGranularityRecommended))
  cuda_assert((hipDeviceGetAttribute)(&UnifiedAddressing, hipDeviceAttributeUnifiedAddressing, Device))
  cuda_assert((hipCtxPushCurrent)(Ctx))

  std::cout << std::boolalpha;
  std::cout << "Api Version: " << ApiVersion << std::endl;
  std::cout << "Unified Addressing: " << (bool)UnifiedAddressing << std::endl;



  RangeSize = align_to(RangeSize, Alignment);

  cuda_assert((hipMemCreate)(&Handle, RangeSize, &AllocationProps, 0))
  cuda_assert((hipMemAddressReserve)(&Address, RangeSize, Alignment, 0, 0))
  cuda_assert((hipMemMap)(Address, RangeSize, 0, Handle, 0))

  std::cout << "No Access: \n";
  std::cout << ptr_attributes(Address) << std::endl;

  hipMemAccessDesc DeviceAccess;
  DeviceAccess.location = AllocationProps.location;
  DeviceAccess.flags = hipMemAccessFlagsProtReadWrite;

  cuda_assert((hipMemSetAccess)(Address, RangeSize, &DeviceAccess, 1))

  std::cout << "\n\nDevice access: \n";
  std::cout << ptr_attributes(Address) << std::endl;


  /*DeviceAccess.location.id = hipCpuDeviceId;
  DeviceAccess.flags = hipMemAccessFlagsProtRead;

  cuda_assert((hipMemSetAccess)(Address, RangeSize, &DeviceAccess, 1))

  std::cout << "\n\nHost access: \n";
  std::cout << ptr_attributes(Address) << std::endl;*/
  /*hipMemAccessDesc BothAccess[2];
  BothAccess[0] = DeviceAccess;
  BothAccess[1].location.type = hipMemLocationTypeDevice;
  BothAccess[1].location.id = Device;
  BothAccess[1].flags = hipMemAccessFlagsProtReadWrite;

  cuda_assert((hipMemSetAccess)(Address, RangeSize, BothAccess, std::size(BothAccess)))

  std::cout << "\n\nBoth access: \n";
  std::cout << ptr_attributes(Address) << std::endl;*/


  hipDeviceptr_t UnifiedDeviceMemory;
  cuda_assert((hipMalloc)(&UnifiedDeviceMemory, RangeSize))

  std::cout << "\n\nUnifiedDeviceMemory: \n";
  std::cout << ptr_attributes(UnifiedDeviceMemory) << std::endl;

  hipDeviceptr_t UnifiedHostMemory;
  cuda_assert((hipHostAlloc)((void**)&UnifiedHostMemory, RangeSize, hipHostMallocMapped | hipHostMallocPortable))

  std::cout << "\n\nUnifiedHostMemory: \n";
  std::cout << ptr_attributes(UnifiedHostMemory) << std::endl;

  hipDeviceptr_t GlobalManagedMemory;
  cuda_assert((hipMallocManaged)(&GlobalManagedMemory, RangeSize, hipMemAttachGlobal))
  hipDeviceptr_t HostManagedMemory;
  cuda_assert((hipMallocManaged)(&HostManagedMemory, RangeSize, hipMemAttachHost))


  std::cout << "\n\nGlobalManagedMemory: \n";
  std::cout << ptr_attributes(GlobalManagedMemory) << std::endl;
  std::cout << "\n\nHostManagedMemory: \n";
  std::cout << ptr_attributes(HostManagedMemory) << std::endl;


  hipDeviceptr_t RegisteredHostMem = (hipDeviceptr_t)malloc(RangeSize);
  assert(RegisteredHostMem);
  cuda_assert((hipHostRegister)((void*)RegisteredHostMem, RangeSize, hipHostRegisterMapped))


  std::cout << "\n\nRegisteredHostMemory: \n";
  std::cout << ptr_attributes(RegisteredHostMem) << std::endl;

  //
  {
    hipStream_t Stream;
    cuda_assert((hipStreamCreateWithFlags)(&Stream, hipStreamNonBlocking))

    cuda_assert((hipLaunchHostFunc)(Stream, [](void* pUserData){
              const char Message[] = "Hello, my name is Maxwell Iverson and I am your captain for this flight, I do hope we can become good friends soon.";
              std::copy(std::execution::par_unseq, std::begin(Message), std::end(Message), (char*)pUserData);
    }, (void*)RegisteredHostMem))
    cuda_assert((cuMemcpyAsync)(UnifiedDeviceMemory, RegisteredHostMem, RangeSize, Stream))
    twiddle_thumbs<<<1, 1, 0, Stream>>>((char*)UnifiedDeviceMemory);
    cuda_assert((cuMemcpyAsync)(RegisteredHostMem, UnifiedDeviceMemory, RangeSize, Stream))
    std::cout << (const char*)RegisteredHostMem << "\n\n";
    cu::sync(Stream);
    std::cout << (const char*)RegisteredHostMem << "\n\n";

    cuda_assert((hipStreamDestroy)(Stream))
  }



  cuda_assert((hipMemUnmap)(Address, RangeSize))
  cuda_assert((hipMemAddressFree)(Address, RangeSize))
  cuda_assert((hipMemRelease)(Handle))

  cuda_assert((hipFree)(UnifiedDeviceMemory))
  cuda_assert((hipFree)(GlobalManagedMemory))
  cuda_assert((hipFree)(HostManagedMemory))
  cuda_assert((hipHostFree)((void*)UnifiedHostMemory))
  cuda_assert((hipHostUnregister)((void*)RegisteredHostMem))

  cuda_assert((hipCtxPopCurrent)(&Ctx))
  cuda_assert((hipDevicePrimaryCtxRelease)(Device))


  free((void*)RegisteredHostMem);

  //print_hello_world<<<1, 64>>>();
  //hipDeviceSynchronize();
  return 0;
}
